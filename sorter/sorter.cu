#ifdef GLAD_SUPPORT
#include <glad/glad.h>
#else
#include <GL/glew.h>
#endif

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include "sorter.h"

void sortByKey(unsigned mortonBuffer, unsigned triangleCount, unsigned indicesBuffer) {
    hipGraphicsResource *mortonRes;
    hipGraphicsResource *indicesRes;
    unsigned * devPtr;
    unsigned * idcPtr;
    size_t size;

    hipGraphicsGLRegisterBuffer(&mortonRes, mortonBuffer, cudaGraphicsMapFlagsNone);
    hipGraphicsMapResources(1, &mortonRes, 0);
    hipGraphicsResourceGetMappedPointer((void **)&devPtr, &size, mortonRes);

    hipGraphicsGLRegisterBuffer(&indicesRes, indicesBuffer, cudaGraphicsMapFlagsNone);
    hipGraphicsMapResources(1, &indicesRes, 0);
    hipGraphicsResourceGetMappedPointer((void **)&idcPtr, &size, indicesRes);

    thrust::device_ptr<unsigned> dptr = thrust::device_pointer_cast(devPtr);
    thrust::device_ptr<unsigned> iptr = thrust::device_pointer_cast(idcPtr);
    thrust::sort_by_key(dptr, dptr + triangleCount, iptr);

    hipGraphicsUnmapResources(1, &mortonRes, 0);
    hipGraphicsUnregisterResource(mortonRes);

    hipGraphicsUnmapResources(1, &indicesRes, 0);
    hipGraphicsUnregisterResource(indicesRes);
}

void sortPair(unsigned mortonBuffer, unsigned triangleCount){
    hipGraphicsResource *mortonRes;
    uint64_t * devPtr;
    size_t size;
    hipGraphicsGLRegisterBuffer(&mortonRes, mortonBuffer, cudaGraphicsMapFlagsNone);
    hipGraphicsMapResources(1, &mortonRes, 0);
    hipGraphicsResourceGetMappedPointer((void **)&devPtr, &size, mortonRes);
    thrust::device_ptr<uint64_t> dptr = thrust::device_pointer_cast(devPtr);
    thrust::sort(dptr, dptr + triangleCount);
    hipGraphicsUnmapResources(1, &mortonRes, 0);
    hipGraphicsUnregisterResource(mortonRes);
}

void sortIndices(unsigned mortonBuffer, unsigned triangleCount) {
    hipGraphicsResource *mortonRes;
    unsigned * devPtr;
    size_t size;
    hipGraphicsGLRegisterBuffer(&mortonRes, mortonBuffer, cudaGraphicsMapFlagsNone);
    hipGraphicsMapResources(1, &mortonRes, 0);
    hipGraphicsResourceGetMappedPointer((void **)&devPtr, &size, mortonRes);
    thrust::device_ptr<unsigned> dptr = thrust::device_pointer_cast(devPtr);
    thrust::sort(dptr, dptr + triangleCount);
    hipGraphicsUnmapResources(1, &mortonRes, 0);
    hipGraphicsUnregisterResource(mortonRes);
}