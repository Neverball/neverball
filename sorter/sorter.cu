#include <GL/glew.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include "sorter.h"

void sortPair(unsigned mortonBuffer, unsigned triangleCount){
    hipGraphicsResource *mortonRes;
    uint64_t * devPtr;
    size_t size;
    hipGraphicsGLRegisterBuffer(&mortonRes, mortonBuffer, cudaGraphicsMapFlagsNone);
    hipGraphicsMapResources(1, &mortonRes, 0);
    hipGraphicsResourceGetMappedPointer((void **)&devPtr, &size, mortonRes);
    thrust::device_ptr<uint64_t> dptr = thrust::device_pointer_cast(devPtr);
    thrust::sort(dptr, dptr + triangleCount);
    hipGraphicsUnmapResources(1, &mortonRes, 0);
    hipGraphicsUnregisterResource(mortonRes);
}

void sortIndices(unsigned mortonBuffer, unsigned triangleCount) {
    hipGraphicsResource *mortonRes;
    unsigned * devPtr;
    size_t size;
    hipGraphicsGLRegisterBuffer(&mortonRes, mortonBuffer, cudaGraphicsMapFlagsNone);
    hipGraphicsMapResources(1, &mortonRes, 0);
    hipGraphicsResourceGetMappedPointer((void **)&devPtr, &size, mortonRes);
    thrust::device_ptr<unsigned> dptr = thrust::device_pointer_cast(devPtr);
    thrust::sort(dptr, dptr + triangleCount);
    hipGraphicsUnmapResources(1, &mortonRes, 0);
    hipGraphicsUnregisterResource(mortonRes);
}